#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include "Signal.h"
#include <complex>
// #include <matplot/matplot.h>



int main(int argc, char* argv[]) {
	hipfftHandle planr2c;
	hipStream_t stream = NULL;

	int batchSize = 4;
	int fftSize = 500;
	int signalLength = batchSize * fftSize;

	using scalar_type = float;
	using inputType = scalar_type;
	using outputType = std::complex<scalar_type>;

	std::vector<inputType> fftInput = generateSignal(50, 100, 200, 0.01, signalLength);
	std::vector<outputType> fftOutput((fftSize / 2 + 1) * batchSize);

	// Print out the first five elements of the input Array
	for (int i = 0; i < 5; i++) {
		std::printf("%f ", fftInput[i]);
	}

	inputType* d_input = nullptr;
	hipfftComplex* d_output = nullptr;

	hipfftCreate(&planr2c);
	hipfftPlan1d(&planr2c, fftSize, HIPFFT_R2C, batchSize);
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	hipfftSetStream(planr2c, stream);


	// Create Device Arrays
	hipMalloc(reinterpret_cast<void**> (&d_input), sizeof(inputType) * fftInput.size());

	hipMalloc(reinterpret_cast<void**>(&d_output), sizeof(outputType) * fftOutput.size());

	hipMemcpyAsync(d_input, fftInput.data(), sizeof(inputType) * fftInput.size(), hipMemcpyHostToDevice, stream);

	// FOrward Transform

	hipfftExecR2C(planr2c, d_input, d_output);

	//Copy data back to the host (CPU)
	hipMemcpyAsync(fftOutput.data(), d_output, sizeof(outputType) * fftOutput.size(), hipMemcpyDeviceToHost, stream);

	hipStreamSynchronize(stream);

	std::printf("Output Array after Forward FFT: \n");

	for (int i = 0; i < 5; i++) {
		std::printf("%f + %fj\n", fftOutput[i].real(), fftOutput[i].imag());
	}

	std::printf("=======================\n");

	hipFree(d_input);
	hipFree(d_output);
	
	hipfftDestroy(planr2c);
	hipStreamDestroy(stream);

	hipDeviceReset();

	return EXIT_SUCCESS;

}